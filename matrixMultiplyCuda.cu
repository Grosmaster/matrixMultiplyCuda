#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

#define BLOCK_SIZE  32
#define N           3200        

__global__ void matMult(float* a, float* b, int n, float* c)
{

    int   bx = blockIdx.x;    
    int   by = blockIdx.y;
    int   tx = threadIdx.x;        
    int   ty = threadIdx.y;
    float sum = 0.0f;           
    int   ia = n * BLOCK_SIZE * by + n * ty;   
    int   ib = BLOCK_SIZE * bx + tx;

    
    for (int k = 0; k < n; k++)
        sum += a[ia + k] * b[ib + k * n];

    int ic = n * BLOCK_SIZE * by + BLOCK_SIZE * bx;

    c[ic + n * ty + tx] = sum;
}


int main(int argc, char* argv[])
{
    printf("START\n");
    int numBytes = N * N * sizeof(float);

    float* a = new float[N * N];
    float* b = new float[N * N];
    float* c = new float[N * N];

    for (int i = 0; i < N; i++)
        for (int j = 0; j < N; j++)
        {
            int	k = N * i + j;
            a[k] = 1.0f;
            b[k] = 1.0f;
        }

    float* adev = NULL;
    float* bdev = NULL;
    float* cdev = NULL;

    hipMalloc((void**)&adev, numBytes);
    hipMalloc((void**)&bdev, numBytes);
    hipMalloc((void**)&cdev, numBytes);

    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocks(N / threads.x, N / threads.y);


    hipMemcpy(adev, a, numBytes, hipMemcpyHostToDevice);
    hipMemcpy(bdev, b, numBytes, hipMemcpyHostToDevice);

    matMult << <blocks, threads >> > (adev, bdev, N, cdev);

    hipMemcpy(c, cdev, numBytes, hipMemcpyDeviceToHost);

    printf("END %d", c[1]);

    hipFree(adev);
    hipFree(bdev);
    hipFree(cdev);

    delete a;
    delete b;
    delete c;

    return 0;
}
